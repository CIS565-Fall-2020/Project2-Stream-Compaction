#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
			int *dev_idata, *dev_odata;
			hipMalloc((void**)&dev_idata, n * sizeof(int));
			hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
			hipMalloc((void**)&dev_odata, n * sizeof(int));
			thrust::device_ptr<int> dev_thrust_odata(dev_odata);
			thrust::device_ptr<int> dev_thrust_idata(dev_idata);

            timer().startGpuTimer();
            // TODO use `thrust::exclusive_scan`
            // example: for device_vectors dv_in and dv_out:
            thrust::exclusive_scan(dev_thrust_idata, dev_thrust_idata + n, dev_thrust_odata);
            timer().endGpuTimer();

			hipMemcpy(odata, thrust::raw_pointer_cast(dev_thrust_odata), n * sizeof(int), hipMemcpyDeviceToHost);
        }
    }
}
