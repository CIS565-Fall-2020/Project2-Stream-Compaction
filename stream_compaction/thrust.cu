#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/copy.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            thrust::host_vector<int> host_i(idata, idata + n);
            thrust::host_vector<int> host_o(n);
            thrust::device_vector<int> dev_i = host_i;
            thrust::device_vector<int> dev_o = host_o;

            timer().startGpuTimer();
            // TODO use `thrust::exclusive_scan`
            // example: for device_vectors dv_in and dv_out:
            // thrust::exclusive_scan(dv_in.begin(), dv_in.end(), dv_out.begin());
            thrust::exclusive_scan(dev_i.begin(), dev_i.end(), dev_o.begin());
            timer().endGpuTimer();

            thrust::copy(dev_o.begin(), dev_o.end(), odata);
        }
    }
}
