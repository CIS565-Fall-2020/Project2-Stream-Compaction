#include "efficient.h"
#include <>
#include <cassert> 
#include "radixSort.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>

int k_th_bit(int k, int n) {
    return (n >> k) & 1;
}

void myprint(int n, int* a, bool abridged = false) {
    printf("    [ ");
    for (int i = 0; i < n; i++) {
        if (abridged && i + 2 == 15 && n > 16) {
            i = n - 2;
            printf("... ");
        }
        printf("%3d ", a[i]);
    }
    printf("]\n");
}

namespace StreamCompaction {
    namespace RadixSort {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        void CpuStandardSort(const int& N, int* out, const int* in) {
            if (N == 0) {
                return;
            }
            assert(in != nullptr);
            assert(out != nullptr);

            std::vector<int> a_vec(in, in + N);

            timer().startCpuTimer();
            std::sort(a_vec.begin(), a_vec.end());
            timer().endCpuTimer();

            std::copy(a_vec.begin(), a_vec.end(), out);
        }

        void GpuRadixSort(const int& N, int* hst_out, const int* hst_in, const int max_bit ){
            //
            if (N == 0) {
                return;
            }
            assert(hst_in != nullptr);
            assert(hst_out != nullptr);

            /*int* dev_in, dev_out, dev_out_buf;
            hipMalloc((void**)&dev_in, N * sizeof(int));
            hipMalloc((void**)&dev_out, N * sizeof(int));
            hipMalloc((void**)&dev_out_buf, N * sizeof(int));
            hipMemcpy(dev_in, hst_in, N * sizeof(int), hipMemcpyHostToDevice);*/

            int* hst_e,* hst_f,* hst_d;
            int* hst_out_buf;
            hst_e = new int[N];
            hst_f = new int[N];
            hst_d = new int[N];

            hst_out_buf = new int[N];
            std::copy(hst_in, hst_in + N, hst_out_buf);

            timer().startGpuTimer();
            for (int k = max_bit; k > 0; k--) {
                for (int i = 0; i < N; i++) {
                    hst_e[i] = 1 - k_th_bit(k-1, hst_out_buf[i]);
                }

                std::cout << "hst_out_buf: ";
                myprint(N, hst_out_buf);

                Efficient::scan(N, hst_f, hst_e, false, false, true);

                int total_falses = hst_e[N - 1] + hst_f[N - 1];
                for (int i = 0; i < N; i++) {
                    hst_d[i] = hst_e[i] == 0 ? (i - hst_f[i] + total_falses) : hst_f[i];
                }

                for (int i = 0; i < N; i++) {
                    hst_out[i] = hst_out_buf[hst_d[i]];
                }
                std::copy(hst_out, hst_out + N, hst_out_buf);

                std::cout << "hst_out: ";
                myprint(N, hst_out);
            }

            timer().endGpuTimer();

            delete[] hst_e;
            delete[] hst_f;
            delete[] hst_d;
            delete[] hst_out_buf;
        }
	}
}
