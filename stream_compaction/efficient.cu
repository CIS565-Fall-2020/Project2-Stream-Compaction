#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"
#include <>
#include <cassert> 
//#include "cis565_stream_compaction_test/testing_helpers.hpp"


namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        __global__ void kernUpdateArray(int idx, int val, int* d_data) {
            d_data[idx] = val;
        }

        __global__ void kernUpSweepStep(
            int N,
            int d_2,
            int* d_data
        ){
            int k = (blockIdx.x * blockDim.x) + threadIdx.x;
            if (k >= N) {
                return;
            }
            if (k % (2 * d_2) == 0) {
                d_data[k + 2 * d_2 - 1] += d_data[k + d_2 - 1];
            }
        }

        __global__ void kernDownSweepStep(
            int N,
            int d_2,
            int* d_data
        ) {
            int k = (blockIdx.x * blockDim.x) + threadIdx.x;
            if (k >= N) {
                return;
            }

            if (k % ( d_2 * 2 )== 0) {
                int tmp = d_data[k + d_2 -1];
                d_data[k + d_2 - 1] = d_data[k + 2 * d_2 - 1];
                d_data[k + 2 * d_2 - 1] = tmp + d_data[k + 2 * d_2 - 1];
            }
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata, bool ifTimer = true) {
            if (n == 0) {
                return;
            }
            assert(odata != nullptr);
            assert(idata != nullptr);

            int log_n = ilog2ceil(n);
            int n_2 = 1 << log_n;

            int* dev_idata;
            dim3 blocksPerGrid = (n_2 + efficient_blocksize - 1) / efficient_blocksize;
            /*int* dev_odata;*/
            hipMalloc((void**)&dev_idata, n_2 * sizeof(int));
            /*hipMalloc((void**)&dev_odata, n_2 * sizeof(int));*/
            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);

            if (ifTimer) {
                timer().startGpuTimer();
            }
            
            // TODO
            for (int d = 0; d <= log_n - 1; d ++) {
                kernUpSweepStep<<<blocksPerGrid, efficient_blocksize >>>(n_2, 1 << d, dev_idata);
            }

            //hipMemcpy(odata, dev_idata, n * sizeof(int), hipMemcpyDeviceToHost);
            kernUpdateArray<<<1, 1>>>(n_2 - 1, 0, dev_idata);
            //hipMemcpy(odata, dev_idata, n * sizeof(int), hipMemcpyDeviceToHost);
            

            for (int d = log_n - 1; d >= 0; d--) {
                kernDownSweepStep << <blocksPerGrid, efficient_blocksize >> > (n_2, 1 << d, dev_idata);
            }

            if (ifTimer) {
                timer().endGpuTimer();
            }
            
            hipMemcpy(odata, dev_idata,n * sizeof(int), hipMemcpyDeviceToHost);
            hipFree(dev_idata);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int N, int *odata, const int *idata) {
            if (N == 0) {
                return 0;
            }
            assert(odata != nullptr);
            assert(idata != nullptr);
            
            int* dev_idata;
            int* dev_odata;
            int* dev_bools;
            int* dev_indices;

            hipMalloc((void**)&dev_idata, N * sizeof(int));
            hipMalloc((void**)&dev_odata, N * sizeof(int));
            hipMalloc((void**)&dev_bools, N * sizeof(int));
            hipMalloc((void**)&dev_indices, N * sizeof(int));

            hipMemcpy(dev_idata, idata, N * sizeof(int), hipMemcpyHostToDevice);

            timer().startGpuTimer();
            // TODO
            dim3 blocksPerGrid = (N + efficient_blocksize - 1) / efficient_blocksize;
            Common::kernMapToBoolean << <blocksPerGrid, efficient_blocksize >> > (N, dev_bools, dev_idata);
            
            scan(N, dev_indices, dev_bools, false);

            Common::kernScatter << <blocksPerGrid, efficient_blocksize >> > (
                N, 
                dev_odata, 
                dev_idata,
                dev_bools,
                dev_indices
            );


            timer().endGpuTimer();

            hipMemcpy(odata, dev_odata, N * sizeof(int), hipMemcpyDeviceToHost);
            int* hst_bools = new int[N];
            hipMemcpy(hst_bools, dev_bools, N * sizeof(int), hipMemcpyDeviceToHost);
            int out = 0;
            for (int i = 0; i < N; i++) {
                if (hst_bools[i] == 1) {
                    out++;
                }
            }

            hipFree(dev_idata);
            hipFree(dev_odata);
            hipFree(dev_bools);
            hipFree(dev_indices);

            return out;
        }
    }

    __global__ void prescan(float* g_odata, float* g_idata, int n) {
        extern __shared__ float temp[];  // allocated on invocation 
        int thid = threadIdx.x; int offset = 1; 
        temp[2 * thid] = g_idata[2 * thid]; // load input into shared memory 
        temp[2*thid+1] = g_idata[2*thid+1]; 

        for (int d = n >> 1; d > 0; d >>= 1)                    // build sum in place up the tree 
        { 
            __syncthreads(); 
            if (thid < d)    
            { 
                int ai = offset * (2 * thid + 1) - 1; 
                int bi = offset * (2 * thid + 2) - 1;
                temp[bi] += temp[ai];
            }    
            offset *= 2;
        }


        if (thid == 0) { temp[n - 1] = 0; } // clear the last element  

        for (int d = 1; d < n; d *= 2) // traverse down tree & build scan 
        {      
            offset >>= 1;      
            __syncthreads();      
            if (thid < d){ 
                int ai = offset * (2 * thid + 1) - 1;     
                int bi = offset * (2 * thid + 2) - 1;

                float t = temp[ai]; 
                temp[ai] = temp[bi]; 
                temp[bi] += t;       
            } 
        }  
        __syncthreads();

        g_odata[2 * thid] = temp[2 * thid]; // write results to device memory      g_odata[2*thid+1] = temp[2*thid+1]; 

    }
}
