#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
		
		// n: number of blocks that need to be swept
		// scaleIndex: 2^(d + 1)
		// offsetLeft: 2^(d) - 1
		// offsetRight: 2^(d + 1) - 1
		__global__ void kernUpSweep(int* oData, int nSwept, int scaleIndex, int offsetLeft, int offsetRight)
		{
			int index = blockIdx.x * blockDim.x + threadIdx.x;
			if (index >= nSwept)
			{
				return;
			}
			int k = index * scaleIndex;
			oData[k + offsetRight] += oData[k + offsetLeft];
		}

		// n: number of blocks that need to be swept
		// scaleIndex: 2^(d + 1)
		// offsetLeft: 2^(d) - 1
		// offsetRight: 2^(d + 1) - 1
		__global__ void kernDownSweep(int* oData, int nSwept, int scaleIndex, int offsetLeft, int offsetRight)
		{
			int index = blockIdx.x * blockDim.x + threadIdx.x;
			if (index >= nSwept)
			{
				return;
			}
			int k = index * scaleIndex;
			int t = oData[k + offsetLeft];
			oData[k + offsetLeft] = oData[k + offsetRight];
			oData[k + offsetRight] += t;
		}

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {

			int *dev_odata;
			int level = ilog2ceil(n);
			int nPOT = 1 << level;	// Clamp n to power-of-two
			hipMalloc((void**)&dev_odata, nPOT * sizeof(int));
			checkCUDAErrorFn("hipMalloc dev_odata1 failed!");
			hipMemset(dev_odata, 0, nPOT * sizeof(int));
			hipMemcpy(dev_odata, idata, n * sizeof(int), hipMemcpyHostToDevice);
			

            timer().startGpuTimer();
            // DONE
			
			// Up Sweep
			int nSwept = nPOT;
			for (int d = 0; d < level; ++d)
			{
				nSwept /= 2;
				dim3 blocksPerGrid((nSwept + threadsPerBlock - 1) / threadsPerBlock);
				int scaleIndex = 1 << (d + 1);
				int offsetLeft = (1 << d) - 1;
				int offsetRight = (1 << (d + 1)) - 1;
				kernUpSweep << <blocksPerGrid, threadsPerBlock >> > (dev_odata, nSwept, scaleIndex, offsetLeft, offsetRight);
			}
			// Set root to zero
			hipMemset(dev_odata + nPOT - 1, 0, sizeof(int));
			// Down Sweep
			nSwept = 1;
			for (int d = level - 1; d >= 0; --d)
			{
				dim3 blocksPerGrid((nSwept + threadsPerBlock - 1) / threadsPerBlock);
				int scaleIndex = 1 << (d + 1);
				int offsetLeft = (1 << d) - 1;
				int offsetRight = (1 << (d + 1)) - 1;
				kernDownSweep << < blocksPerGrid, threadsPerBlock >> > (dev_odata, nSwept, scaleIndex, offsetLeft, offsetRight);
				nSwept *= 2;
			}

            timer().endGpuTimer();

			hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);
			hipFree(dev_odata);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {

			int *dev_bools;
			int *dev_indices;
			int *dev_idata;

			int level = ilog2ceil(n);
			int nPOT = 1 << level;	// Clamp n to power-of-two

			hipMalloc((void**)&dev_bools, nPOT * sizeof(int));
			checkCUDAErrorFn("hipMalloc dev_bools failed!");
			hipMalloc((void**)&dev_indices, nPOT * sizeof(int));
			checkCUDAErrorFn("hipMalloc dev_indices failed!");
			hipMalloc((void**)&dev_idata, nPOT * sizeof(int));
			checkCUDAErrorFn("hipMalloc dev_idata failed!");

			hipMemset(dev_idata, 0, nPOT * sizeof(int));
			hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);


            timer().startGpuTimer();
            // DONE
			// Step1: Map idata to bools
			dim3 blocksPerGridnPOT((nPOT + threadsPerBlock - 1) / threadsPerBlock);
			Common::kernMapToBoolean << <blocksPerGridnPOT, threadsPerBlock >> > (nPOT, dev_bools, dev_idata);
			hipMemcpy(dev_indices, dev_bools, nPOT * sizeof(int), hipMemcpyDeviceToDevice);
			// Step2: Scan indices
			// Up Sweep
			int nSwept = nPOT;
			for (int d = 0; d < level; ++d)
			{
				nSwept /= 2;
				dim3 blocksPerGrid((nSwept + threadsPerBlock - 1) / threadsPerBlock);
				int scaleIndex = 1 << (d + 1);
				int offsetLeft = (1 << d) - 1;
				int offsetRight = (1 << (d + 1)) - 1;
				kernUpSweep << <blocksPerGrid, threadsPerBlock >> > (dev_indices, nSwept, scaleIndex, offsetLeft, offsetRight);
			}
			// Set root to zero
			hipMemset(dev_indices + nPOT - 1, 0, sizeof(int));
			// Down Sweep
			nSwept = 1;
			for (int d = level - 1; d >= 0; --d)
			{
				dim3 blocksPerGrid((nSwept + threadsPerBlock - 1) / threadsPerBlock);
				int scaleIndex = 1 << (d + 1);
				int offsetLeft = (1 << d) - 1;
				int offsetRight = (1 << (d + 1)) - 1;
				kernDownSweep << < blocksPerGrid, threadsPerBlock >> > (dev_indices, nSwept, scaleIndex, offsetLeft, offsetRight);
				nSwept *= 2;
			}
			// Step3: Scatter
			Common::kernScatter << <blocksPerGridnPOT, threadsPerBlock >> > (nPOT, dev_idata, dev_idata, dev_bools, dev_indices);

            timer().endGpuTimer();
			hipMemcpy(odata, dev_idata, n * sizeof(int), hipMemcpyDeviceToHost);

			int lastIndex = 0;
			hipMemcpy(&lastIndex, dev_indices + nPOT - 1, sizeof(int), hipMemcpyDeviceToHost);
			int lastBool = 0;
			hipMemcpy(&lastBool, dev_bools + nPOT - 1, sizeof(int), hipMemcpyDeviceToHost);

			hipFree(dev_bools);
			hipFree(dev_indices);
			hipFree(dev_idata);
            return lastIndex + lastBool;
        }
    }
}
