#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }


        __global__ void prescan(int n, float* g_odata, float* g_idata) {
            extern __shared__ float temp[];
            // allocated on invocation 
            int thid = threadIdx.x;
            int offset = 1;
            temp[2 * thid] = g_idata[2 * thid]; // load input into shared memory
            temp[2*thid+1] = g_idata[2*thid+1];

            // build sum in place up the tree
            for (int d = n >> 1; d > 0; d >>= 1) {
                __syncthreads();    
                if (thid < d) {
                    int ai = offset * (2 * thid + 1) - 1;
                    int bi = offset * (2 * thid + 2) - 1;
                    temp[bi] += temp[ai];
                }
                offset *= 2;
            }
            if (thid == 0) { temp[n - 1] = 0; } // clear the last element
            
            // traverse down tree & build scan
            for (int d = 1; d < n; d *= 2) {
                offset >>= 1;
                __syncthreads();
                if (thid < d) {
                    int ai = offset * (2 * thid + 1) - 1;
                    int bi = offset * (2 * thid + 2) - 1;
                    float t = temp[ai];
                    temp[ai] = temp[bi];
                    temp[bi] += t;
                }
            }
            __syncthreads();
            g_odata[2 * thid] = temp[2 * thid];
            // write results to device memory
            g_odata[2*thid+1] = temp[2*thid+1];
        }

        __device__ void kernUpSweep() {

        }

        __device__ void kernDownSweep() {

        }

        __global__ void kernExScan() {
            int idx = threadIdx.x + (blockIdx.x * blockDim.x);


        }


        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {

            return;

            int* dev_idata;
            hipMalloc((void**)&dev_idata, n * sizeof(int));
            checkCUDAError("hipMalloc dev_idata failed!");

            hipMemcpy(dev_idata, idata, sizeof(int) * n, hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy idata to dev_idata failed!");

            // for most gpus there are 1024 threads per block
            int threadsPerBlock = 1024;
            int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock; // ceiling of n / threadsPerBlock
            dim3 blockDim(threadsPerBlock, 0, 0);
            dim3 gridDim(blocksPerGrid, 0, 0);


            timer().startGpuTimer();
            // TODO
            int k = ilog2ceil(n);
            // kernScan << <gridDim, blockDim >> > ();

            timer().endGpuTimer();

            hipFree(dev_idata);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        using namespace StreamCompaction::Common;
        int compact(int n, int *odata, const int *idata) {
            return -1;

            int* dev_idata;
            int* dev_odata;
            bool* dev_bools;
            int* dev_indices;

            hipMalloc((void**)&dev_idata, n * sizeof(int));
            checkCUDAError("hipMalloc dev_idata failed!");

            hipMemcpy(dev_idata, idata, sizeof(int) * n, hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy idata to dev_idata failed!");

            hipMalloc((void**)&dev_odata, n * sizeof(int));
            checkCUDAError("hipMalloc dev_odata failed!");

            hipMalloc((void**)&dev_bools, n * sizeof(bool));
            checkCUDAError("hipMalloc dev_mask failed!");

            hipMalloc((void**)&dev_indices, n * sizeof(bool));
            checkCUDAError("hipMalloc dev_indices failed!");

            // for most gpus there are 1024 threads per block
            int threadsPerBlock = 1024;
            int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock; // ceiling of n / threadsPerBlock
            dim3 blockDim(threadsPerBlock, 0, 0);
            dim3 gridDim(blocksPerGrid, 0, 0);


            timer().startGpuTimer();
            // TODO
            int k = ilog2ceil(n);
            // step 1: compute dev_bools = determine which elements should be purged
            // kernMapToBoolean<<<gridDim, blockDim>>>(n, dev_bools, dev_idata);
            // step 2: exclusive scan on dev_bools
            // kernScan<<<gridDim, blockDim>>>(n, dev_indices, dev_bools);
            // step 3: reduce the array based on bools
            // kernScatter<<<gridDim, blockDim>>>(n, dev_odata, dev_idata, dev_bools, dev_indices);

            timer().endGpuTimer();

            hipFree(dev_idata);
            hipFree(dev_odata);
            hipFree(dev_bools);
            hipFree(dev_indices);

            return -1;
        }
    }
}
