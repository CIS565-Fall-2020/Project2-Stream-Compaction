#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }


        __global__ void prescan(float* g_odata, float* g_idata, int n) {
            extern __shared__ float temp[];
            // allocated on invocation 
            int thid = threadIdx.x; int offset = 1;
        }


        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            int* dev_idata;
            hipMalloc((void**)&dev_idata, n * sizeof(int));
            checkCUDAError("hipMalloc dev_idata failed!");

            hipMemcpy(dev_idata, idata, sizeof(int) * n, hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy idata to dev_idata failed!");

            // for most gpus there are 1024 threads per block
            int threadsPerBlock = 1024;
            int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock; // ceiling of n / threadsPerBlock
            dim3 blockDim(threadsPerBlock, 0, 0);
            dim3 gridDim(blocksPerGrid, 0, 0);


            timer().startGpuTimer();
            // TODO
            int k = ilog2ceil(n);
            kernScan << <gridDim, blockDim >> > ();

            timer().endGpuTimer();

            hipFree(dev_idata);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            // TODO
            timer().endGpuTimer();
            return -1;
        }
    }
}
