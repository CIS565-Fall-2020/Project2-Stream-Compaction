#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void upSweep(int *data, int d) {
          int idx = threadIdx.x + (blockIdx.x * blockDim.x);
          int interval = 2 << d;
          int mapped = interval * idx + interval - 1;
          data[mapped] += data[mapped - (interval >> 1)];
        }

        __global__ void downSweep(int *data, int d) {
          int idx = threadIdx.x + (blockIdx.x * blockDim.x);
          int interval = 2 << d;
          int node = interval * idx + interval - 1;
          int left = node / 2;
          int temp = data[left];
          data[left] = data[node];
          data[node] += temp;
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *dev_odata, const int *dev_idata) {
          int iterations = ilog2ceil(n);
          int nextN = 2 << iterations;
          int *dev_idata_temp;
          hipMalloc((void **) &dev_idata_temp, nextN * sizeof(int));
          hipMemset(dev_idata_temp, 0, nextN *sizeof(int));
          hipMemcpy(dev_idata_temp, dev_idata, sizeof(int) * n, hipMemcpyDeviceToDevice);
          timer().startGpuTimer();

          // Up-sweep
          for (int d = 1; d <= iterations; d++) {
            int numThreads = 2 << (iterations - d);
            dim3 blocks((numThreads + blockSize - 1) / blockSize);
            upSweep<<<blocks, blockSize>>>(dev_idata_temp, d);
          }

          // Down-sweep
          // Set the "root" to 0
          hipMemset(dev_idata + n - 1, 0, sizeof(int));
          for (int d = iterations; d >= 1; d--) {
            int numThreads = 2 << (iterations - d);
            dim3 blocks((numThreads + blockSize - 1) / blockSize);
            downSweep<<<blocks, blockSize>>>(dev_idata_temp, d);
          }
          
          timer().endGpuTimer();
          hipMemcpy(dev_odata, dev_idata_temp, sizeof(int) * n, hipMemcpyDeviceToDevice);
          hipFree(dev_idata_temp);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
          int *bools, *indices, dev_idata, dev_odata;
          hipMalloc((void**) &bools, sizeof(int) * n);
          hipMalloc((void**) &indices, sizeof(int) * n);
          hipMalloc((void**) &dev_idata, sizeof(int) * n);
          hipMalloc((void**) &dev_odata, sizeof(int) * n);
          hipMemcpy(dev_idata, idata, sizeof(int) * n, hipMemcpyHostToDevice);

          timer().startGpuTimer();

          dim3 blocks((n + blockSize - 1) / blockSize);
          Common::kernMapToBoolean<<<blocks, blockSize>>>(n, bools, dev_idata);
          scan(n, indices, bools);
          Common::kernScatter<<<blocks, blockSize>>>(n, dev_odata, dev_idata, bools, indices);

          timer().endGpuTimer();

          hipMemcpy(odata, dev_odata, sizeof(int) * n, hipMemcpyDeviceToHost);
          hipFree(bools);
          hipFree(indices);
          hipFree(dev_idata);
          hipFree(dev_odata);
          return -1;
        }
    }
}
