#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"
#include <iostream>

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

       __global__ void upSweepEfficient(int n, int offset, int sDataSize, int* data) {
          extern __shared__ int sData[];
          int tIdx = threadIdx.x;
          sData[tIdx] = data[blockIdx.x * blockDim.x + tIdx];
          for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
            if (tIdx < stride) {
              __syncthreads();
              int s1 = sData[2 * tIdx];
              int s2 = sData[2 * tIdx + 1];
              __syncthreads();
              sData[tIdx] = s1 + s2;
            }
          }
          __syncthreads();
          data[blockIdx.x * blockDim.x + tIdx] = sData[sDataSize - tIdx];
        }

        __global__ void upSweep(int n, int offset, int* data) {
          int index = (blockIdx.x * blockDim.x) + threadIdx.x;
          if (index < n && index % offset == 0) {
            data[index + offset - 1] = data[index + offset / 2 - 1] + data[index + offset - 1];
          }
        }

        __global__ void downSweep(int n, int offset, int* data) {
          int index = (blockIdx.x * blockDim.x) + threadIdx.x;
          if (index < n && index % offset == 0) {
            int halfOffset = offset / 2; // Helps to find left child
            int t = data[index + halfOffset - 1];
            // Set right child to be the same as parent's value
            data[index + halfOffset - 1] = data[index + offset - 1];
            // Set left child to be the sum of parent and parent's sibling
            data[index + offset - 1] += t;
          }
        }

        __global__ void setLastElementZero(int n, int* data) {
          data[n - 1] = 0;
        }

        __global__ void formatFinalData(int n, int* odata, const int* idata) {
          int index = (blockIdx.x * blockDim.x) + threadIdx.x;
          if (index >= n) {
            return;
          }
          odata[index] = idata[index];
        }


        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
          // Calculate the number of elements the input can be treated as an array with a power of two elements
          int kernelInvokeCount = ilog2ceil(n);
          int n2 = (int)pow(2, kernelInvokeCount);
          
          // Declare, allocate, and transfer data on gpu from cpu
            int* dev_odata;
            int* dev_odata2;

            hipMalloc((void**)&dev_odata, n2 * sizeof(int));
            checkCUDAError("hipMalloc dev_odata failed!");
            
            hipMalloc((void**)&dev_odata2, n2 * sizeof(int));
            checkCUDAError("hipMalloc dev_odata2 failed!");

            hipMemcpy(dev_odata, idata, sizeof(int) * n, hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy idata to dev_odata failed!");

            if (n < 1) {
              return;
            }
            timer().startGpuTimer();

            int blockSize = 128;
            dim3 blockCount((n2 + blockSize - 1) / blockSize);

            // Format input data (pad 0s to the closest power of two elements, inclusively)
            StreamCompaction::Common::formatInitData << <blockCount, blockSize >> > (n, n2, dev_odata);

            for (int i = 0; i <= kernelInvokeCount; i++) {
              int offset = (int) pow(2, i + 1);
              upSweep << <blockCount, blockSize >> > (n2, offset, dev_odata);
            }

            setLastElementZero << <blockCount, blockSize >> > (n2, dev_odata);

            for (int i = kernelInvokeCount - 1; i >= 0; i--) {
              int offset = (int) pow(2, i + 1);
              downSweep << <blockCount, blockSize >> > (n2, offset, dev_odata);
            }

            formatFinalData << < blockCount, blockSize >> > (n, dev_odata2, dev_odata);

            timer().endGpuTimer();

            // Transfer data from gpu to cpu
            hipMemcpy(odata, dev_odata2, sizeof(int) * n, hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy dev_odata to odata failed!");

            hipFree(dev_odata);
            checkCUDAError("hipFree dev_odata failed!");

            hipFree(dev_odata2);
            checkCUDAError("hipFree dev_odata2 failed!");
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            //timer().startGpuTimer();
            
            // TODO
            
            int blockSize = 128;
            dim3 blockCount((n + blockSize - 1) / blockSize);

            // Declare, allocate memory in GPU and transfer memory from CPU to GPU
            int* dev_idata;
            int* dev_bools;
            int* dev_indices;
            int* dev_odata;

            hipMalloc((void**)&dev_idata, n * sizeof(int));
            checkCUDAError("hipMalloc dev_idata failed!");

            hipMalloc((void**)&dev_bools, n * sizeof(int));
            checkCUDAError("hipMalloc dev_bools failed!");

            hipMalloc((void**)&dev_indices, n * sizeof(int));
            checkCUDAError("hipMalloc dev_indices failed!");

            hipMalloc((void**)&dev_odata, n * sizeof(int));
            checkCUDAError("hipMalloc dev_odata failed!");

            hipMemcpy(dev_idata, idata, sizeof(int) * n, hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy dev_idata failed!");

            // Compute temorary arrray containing
            // 1 if corresponding element meets criteria (of not equal to 0)
            // 0 if element does not meete criteria (of not equal to 0)
            StreamCompaction::Common::kernMapToBoolean << <blockCount, blockSize >> > (n, dev_bools, dev_idata);

            // Run exclusive scan on temporary array
            scan(n, dev_indices, dev_bools);
            StreamCompaction::Common::kernScatter << <blockCount, blockSize >> > (n, dev_odata, dev_idata, dev_bools, dev_indices);
            
            std::unique_ptr<int[]> bools{ new int[n] };
            std::unique_ptr<int[]> indices{ new int[n] };

            hipMemcpy(bools.get(), dev_bools, sizeof(int) * n, hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy dev_bools failed!");

            hipMemcpy(indices.get(), dev_indices, sizeof(int) * n, hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy dev_bools failed!");

            hipMemcpy(odata, dev_odata, sizeof(int) * n, hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy dev_odata failed!");

            // Clean up
            hipFree(dev_idata);
            hipFree(dev_bools);
            hipFree(dev_indices);
            hipFree(dev_odata);

            int remaining = bools[n - 1] == 1? indices[n - 1] : indices[n - 1] - 1;
            remaining++; 

            //timer().endGpuTimer();
            return remaining;
        }
    }
}
