#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

#include <string>

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        // TODO: __global__
        // This version can handle arrays only as large as can be processed by a single thread block running on one multiprocessor of a GPU.
        //__global__ void scan(float* g_odata, float* g_idata, int n) {
        //    extern __shared__ float temp[]; 
        //    // allocated on invocation    
        //    int thid = threadIdx.x;   int pout = 0, pin = 1;   
        //    // Load input into shared memory.   
        //    // This is exclusive scan, so shift right by one    
        //    // and set first element to 0
        //    temp[pout*n + thid] = (thid > 0) ? g_idata[thid-1] : 0;
        //    __syncthreads(); 
        //    for (int offset = 1; offset < n; offset *= 2)   
        //    {     
        //        pout = 1 - pout; 
        //        // swap double buffer indices     
        //        pin = 1 - pout;     
        //        if (thid >= offset)       
        //            temp[pout*n+thid] += temp[pin*n+thid - offset];     
        //        else       
        //            temp[pout*n+thid] = temp[pin*n+thid];     
        //        __syncthreads();
        //    }   
        //    g_odata[thid] = temp[pout*n+thid]; 
        //    // write output 
        //} 

        __global__ void kernInitExScan(int n, int* temp, int* idata) {
            int idx = threadIdx.x + (blockIdx.x * blockDim.x);
            if (idx >= n)
                idata[idx] = 0;
            // shift the array to the right by one for exclusive scan
            // the initializing the padding of idata inn the above line is not guaranteed to be 
            // completed for all threads by the time the next line is reached
            // so just initialize all of the padding in the temp to 0 here
            temp[idx] = (idx > 0 && idx < n) ? idata[idx - 1] : 0;
        }

        __global__ void kernExScan(int pN, int* temp, int* odata, const int*idata, int offset, int pingpong) {
            int idx = threadIdx.x + (blockIdx.x * blockDim.x);

            if (idx >= offset)
                temp[pingpong * pN + idx] += temp[(1 - pingpong) * pN + idx - offset];
            else
                temp[pingpong * pN + idx] = temp[(1 - pingpong) * pN + idx];
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        using namespace std;
        void scan(int n, int *odata, const int *idata) {
            int* dev_idata;
            int* dev_odata;
            int* dev_temp;

            int depth = ilog2ceil(n);
            // remember numbers are read from right to left
            int pN = 1 << depth;    // n rounded to the next power of 2 = n after padding

            // allocating memory for dev_idata and copying memory over from idata
            hipMalloc((void**)&dev_idata, pN * sizeof(int));
            checkCUDAError("hipMalloc dev_idata failed!");

            // std::unique_ptr<int[]>intKeys{ new int[N] };
            hipMemcpy(dev_idata, idata, sizeof(int) * n, hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy idata to dev_idata failed!");

            // allocating memory for dev_odata
            hipMalloc((void**)&dev_odata, n * sizeof(int));
            checkCUDAError("hipMalloc dev_odata failed!");

            // allocating memory for dev_temp
            hipMalloc((void**)&dev_temp, 2 * pN * sizeof(int));
            checkCUDAError("hipMalloc dev_temp failed!");

            // for most gpus 1024 is the maximum number of threads per block
            int threadsPerBlock = 1024;
            int blocksPerGrid = (pN + threadsPerBlock - 1) / threadsPerBlock; // ceiling of ( pN / threadsPerBlock )
            dim3 blockDim(threadsPerBlock);
            dim3 gridDim(blocksPerGrid);

            timer().startGpuTimer();
            // initializes buffers necessary for naive exclusive scan
            kernInitExScan<<<gridDim, blockDim>>>(n, dev_temp, dev_idata);
            checkCUDAError("kernInitExScan failed!");
            // execution of naive exclusive scan in parallel
            // uses global memory instead of shared memory for ping pong buffers
            // so that the data can be of arbitrary size
            int pingpong = 0;
            for (int offset = 1; offset < pN; offset *= 2) {
                kernExScan<<<gridDim, blockDim>>>(pN, dev_temp, dev_odata, dev_idata, offset, pingpong);
                checkCUDAError("kernExScan failed!");
                pingpong = 1 - pingpong;
            }
            timer().endGpuTimer();

            hipFree(dev_idata);
            hipFree(dev_odata);
            hipFree(dev_temp);
        }
    }
}
