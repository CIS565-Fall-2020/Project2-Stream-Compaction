#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"
#include <memory>
#include <iostream>

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        // TODO: __global__

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */


        /* Copy initial data over and pad 0's if out of scope of initial size 
         * aka the input array has a smaller initial size than the final array, 
         * and anything larger than index [size of input array] will be 0 in the output array
         */
        __global__ void formatInitData(int initSize, int finalSize, int* data) {
          int index = (blockIdx.x * blockDim.x) + threadIdx.x; 
          if (index >= initSize && index < finalSize) {
            data[index] = 0;
          }
        }

        __global__ void add(int n, int ignoreIndexCount, int* odata, const int* idata) {
          int index = (blockIdx.x * blockDim.x) + threadIdx.x;
          if (index < ignoreIndexCount) {
            odata[index] = idata[index];
          } else if (index < n) {
            int x1 = idata[index - ignoreIndexCount];
            int x2 = idata[index];
            odata[index] = x1 + x2;
          }
        }

        // Careful with non-power of 2
        void scan(int n, int *odata, const int *idata) {
            if (n < 1) {
              return;
            }
            // Calculate the number of elements the input can be treated as an array with a power of two elements
            int kernelInvokeCount = ilog2ceil(n);
            int n2 = pow(2, kernelInvokeCount);
            
            // Declare data to be on the gpu
            int* dev_odata;
            int* dev_idata;
            std::unique_ptr<int[]> tdata{ new int[n2] };

            // Allocate data to be on the gpu
            hipMalloc((void**)&dev_odata, n2 * sizeof(int));
            checkCUDAError("hipMalloc dev_odata failed!");
            
            hipMalloc((void**)&dev_idata, n2 * sizeof(int));
            checkCUDAError("hipMalloc dev_tdata failed!");

            // Transfer data from cpu to gpu
            hipMemcpy(dev_idata, idata, sizeof(int) * n, hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy dev_idata failed!");

            timer().startGpuTimer();

            int blockSize = 128;
            dim3 blockCount((n2 + blockSize - 1) / blockSize);

            // Format input data (pad 0s to the closest power of two elements, inclusively)
            StreamCompaction::Common::formatInitData << <blockCount, blockSize >> > (n, n2, dev_idata);

            std::cout << "kernel invoke count: " << kernelInvokeCount << std::endl;

            for (int i = 1; i <= kernelInvokeCount; i++) {
              int ignoreIndexCount = pow(2, i - 1);
              add << <blockCount, blockSize >> > (n2, ignoreIndexCount, dev_odata, dev_idata);

              int* temp = dev_idata;
              dev_idata = dev_odata;
              dev_odata = temp;
            }

            // Shift things to the right to make the inclusive can into exclusive scan
            StreamCompaction::Common::shiftRight<< <blockCount, blockSize >> > (n, dev_idata, dev_odata);

            // Transfer data from gpu to cpu
            hipMemcpy(odata, dev_odata, sizeof(int) * n, hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy dev_odata failed!");

            hipFree(dev_odata);
            checkCUDAError("hipFree dev_odata failed!");
            hipFree(dev_idata);
            checkCUDAError("hipFree dev_idata failed!");
            
            // Calculate the number of blocks and threads per block
            timer().endGpuTimer();
        }
    }
}
