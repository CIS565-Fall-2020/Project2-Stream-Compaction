#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        int* dev_bufferA;
        int* dev_bufferB;
        int numObjects;
        
        __global__ void kernNaiveScan(int N, int* A, int* B, int d) {
            int index = (blockIdx.x * blockDim.x) + threadIdx.x;
            if (index >= N) {
                return;
            }
            int temp = powf(2, d - 1);
            if (index < temp) {
                if (d % 2 == 0) {
                    B[index] = A[index];
                }
                else {
                    A[index] = B[index];
                }
                return;
            }
            if (d % 2 == 0) {
                B[index] = A[index - temp] + A[index];
            }
            else {
                A[index] = B[index - temp] + B[index];
            }
        }

        void initSimulation(int N, const int* B) {
            numObjects = N;
            hipMalloc((void**)&dev_bufferA, N * sizeof(int));
            hipMalloc((void**)&dev_bufferB, N * sizeof(int));
            int a = B[0];
            hipMemcpy(dev_bufferA, &a, sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(dev_bufferB, B, N * sizeof(int), hipMemcpyHostToDevice);

            hipDeviceSynchronize();
        }

        void endSimulation() {
            hipFree(dev_bufferA);
            hipFree(dev_bufferB);
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            
            initSimulation(n, idata);
            const int blockSize = 64;
            timer().startGpuTimer();
            dim3 numBoidBlocks((n + blockSize - 1) / blockSize);
            int dmax = ilog2ceil(n);

            for (int i = 1; i <= dmax; i++) {
                kernNaiveScan << <numBoidBlocks, blockSize >> > (n, dev_bufferA, dev_bufferB, i);
                hipDeviceSynchronize();
                
            }
            if (dmax % 2 == 0) {
                hipMemcpy(odata + 1, dev_bufferB, (n - 1) * sizeof(int), hipMemcpyDeviceToHost);
            }
            else {
                hipMemcpy(odata + 1, dev_bufferA, (n - 1) * sizeof(int), hipMemcpyDeviceToHost);
            }
            odata[0] = 0;
            endSimulation();
            timer().endGpuTimer();
        }
    }
}
