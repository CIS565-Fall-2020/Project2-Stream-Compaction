#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        // TODO: __global__
        __global__ void scan(float* g_odata, float* g_idata, int n) {
            extern __shared__ float temp[]; 
            // allocated on invocation    
            int thid = threadIdx.x;   int pout = 0, pin = 1;   
            // Load input into shared memory.   
            // This is exclusive scan, so shift right by one    
            // and set first element to 0
            temp[pout*n + thid] = (thid > 0) ? g_idata[thid-1] : 0;
            __syncthreads(); 
            for (int offset = 1; offset < n; offset *= 2)   
            {     
                pout = 1 - pout; 
                // swap double buffer indices     
                pin = 1 - pout;     
                if (thid >= offset)       
                    temp[pout*n+thid] += temp[pin*n+thid - offset];     
                else       
                    temp[pout*n+thid] = temp[pin*n+thid];     
                __syncthreads();   
            }   
            g_odata[thid] = temp[pout*n+thid]; 
            // write output 
        } 

        __global__ void kernScan() {
            int id = threadIdx.x + blockIdx.x;
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            int* dev_idata;
            hipMalloc((void**)&dev_idata, n * sizeof(int));
            checkCUDAError("hipMalloc dev_idata failed!");

            hipMemcpy(dev_idata, idata, sizeof(int) * n, hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy idata to dev_idata failed!");

            // for most gpus there are 1024 threads per block
            int threadsPerBlock = 1024;
            int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock; // ceiling of n / threadsPerBlock
            dim3 blockDim(threadsPerBlock, 0, 0);
            dim3 gridDim(blocksPerGrid, 0, 0);

            
            timer().startGpuTimer();
            // TODO
            int k = ilog2ceil(n);
            // kernScan<<<gridDim, blockDim >>>();

            timer().endGpuTimer();

            hipFree(dev_idata);
        }
    }
}
